#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <limits>

#include "CycleTimer.h"
#include "sparse.h"

#define TILE_WIDTH 32
#define TILE_SIZE (TILE_WIDTH * TILE_WIDTH)
#define RM(r, c, width) ((r) * (width) + (c))

__global__ void mm_kernel(int m, int p, int n, float *device_A, float *device_B, float *device_C) {
  // (m, p) * (p, n)
  // blockDim.y == blockDim.x == TILE_WIDTH
  int row_start = blockDim.y * blockIdx.y;
  int col_start = blockDim.y * blockIdx.x;
  int row_offset = threadIdx.y;
  int col_offset = threadIdx.x;
  float res = 0.f;
  __shared__ float A[TILE_WIDTH * TILE_WIDTH];
  __shared__ float B[TILE_WIDTH * TILE_WIDTH];
  for (int offset = 0; offset < p; offset += TILE_WIDTH) {
    int A_row_idx = row_start + row_offset;
    int A_col_idx = offset + col_offset;
    int B_row_idx = offset + row_offset;
    int B_col_idx = col_start + col_offset;
    if (A_row_idx < m && A_col_idx < p) {
      A[row_offset * TILE_WIDTH + col_offset] = device_A[p * A_row_idx + A_col_idx];
    } else {
      A[row_offset * TILE_WIDTH + col_offset] = 0.f;
    }
    if (B_row_idx < p && B_col_idx < n) {
      B[row_offset * TILE_WIDTH + col_offset] = device_B[n * B_row_idx + B_col_idx];
    } else {
      B[row_offset * TILE_WIDTH + col_offset] = 0.f;
    }
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++) {
      res += A[row_offset * TILE_WIDTH + k] * B[k * TILE_WIDTH + col_offset];
    }
    __syncthreads();
  }
  int res_row_idx = row_start + row_offset;
  int res_col_idx = col_start + col_offset;
  if (res_row_idx < m && res_col_idx < n) {
    device_C[res_row_idx * n + res_col_idx] = res;
  }
}

__global__ void heat_kernel(int num_nodes, int num_heads, int out_dim,
                            float *device_msgs, float *device_A, float *device_heats) {
  // device_A: (num_heads, 2, out_dim)
  // device_msgs: (num_nodes, num_heads, out_dim)
  // device_heats: (num_heads, 2, num_nodes)
  // grid: (num_heads, num_nodes / tile_width)
  // block: (2, tile_width)
  extern __shared__ float A[];
  int head_idx = blockIdx.x;
  int block_size = blockDim.x * blockDim.y;
  int idx_within_block = threadIdx.x * blockDim.y + threadIdx.y;
  // load device_A[head_idx, :, :]
  int A_start_idx = head_idx * 2 * out_dim;
  int node_idx = blockDim.y * blockIdx.y + threadIdx.y;
  int num_per_thread = (2 * out_dim + block_size - 1) / block_size;
  for (int k = 0; k < num_per_thread; k++) {
    int local_idx = k + idx_within_block * num_per_thread;
    if (local_idx < 2 * out_dim) {
      A[local_idx] = device_A[A_start_idx + local_idx];
    }
  }
  __syncthreads();
  // device_msgs[node_idx, head_idx, :] * A[threadIdx.x, :]
  float res = 0.f;
  if (node_idx < num_nodes) {
    for (int k = 0; k < out_dim; k++) {
      res += device_msgs[node_idx * num_heads * out_dim + head_idx * out_dim + k] * A[threadIdx.x * out_dim + k];
    }
    // device_heats[head_idx, threadIdx.x, node_idx]
    device_heats[head_idx * 2 * num_nodes + threadIdx.x * num_nodes + node_idx] = res;
  }
}

__global__ void attn_kernel(int num_nodes, int num_elems, int *col_idx, int *delim,
                            float *heats, float *attn, float min_f) {
  // heats: (num_heads, 2, num_nodes)
  // attn: (num_heads, num_elems)
  // col_idx: (num_elems,)
  // delim: (num_nodes,)
  // grid: (num_heads, num_nodes / tile_size)
  // block: (tile_size)
  int head_idx = blockIdx.x;
  int node_idx = blockDim.y * blockIdx.y + threadIdx.x;
  if (node_idx >= num_nodes) {
    return;
  }
  float max_affinity = min_f;
  // heats[head_idx, 0, node_idx]
  float curr_node_heat = heats[head_idx * 2 * num_nodes + node_idx];

  int col_start = delim[node_idx];
  int col_end = delim[node_idx + 1];
  for (int k = col_start; k < col_end; k++) {
    int neighbor_idx = col_idx[k];
    // heats[head_idx, 1, neighbor_idx]
    float neighbor_node_heat = heats[head_idx * 2 * num_nodes + num_nodes + neighbor_idx];
    float heat_sum = curr_node_heat + neighbor_node_heat;
    float curr_affinity = (heat_sum > 0.f) ? heat_sum : (0.2f * heat_sum);
    if (curr_affinity > max_affinity) max_affinity = curr_affinity;
    // attn[head_idx, k]
    attn[head_idx * num_elems + k] = curr_affinity;
  }
  float affinity_sum = 0.f;
  for (int k = col_start; k < col_end; k++) {
    float curr_affinity = exp(attn[head_idx * num_elems + k] - max_affinity);
    affinity_sum += curr_affinity;
    attn[head_idx * num_elems + k] = curr_affinity;
  }
  for (int k = col_start; k < col_end; k++) {
    attn[head_idx * num_elems + k] /= affinity_sum;
  }

}

__global__ void aggregate_kernel(int num_nodes, int num_heads, int out_dim, int num_elems,
                                 float *device_msgs, float *device_attn,
                                 int *device_col_idx, int *device_delim, float *device_output_feats) {
  // grid: (num_heads, (num_nodes * out_dim) / tile_size)
  // block: (tile_size)
  // device_out_feats: (num_nodes, num_head, out_dim)
  // device_msgs: (num_nodes, num_heads, out_dim)
  // device_attn: (num_heads, num_elems)
  // device_col_idx: (num_elems,)
  // device_delim: (num_nodes, )

  int head_idx = blockIdx.x;
  int idx_within_head = blockIdx.y * TILE_SIZE + threadIdx.x;
  int node_idx = idx_within_head / out_dim;
  int local_feat_idx = idx_within_head % out_dim;
  int feat_idx = head_idx * out_dim + local_feat_idx;
  int neighbor_start = device_delim[node_idx];
  int neighbor_end = device_delim[node_idx + 1];
  if (idx_within_head < num_nodes * out_dim) {
    int global_idx = node_idx * num_heads * out_dim + feat_idx;
    device_output_feats[global_idx] = 0.f;
    for (int k = neighbor_start; k < neighbor_end; k++) {
      int neighbor_idx = device_col_idx[k];
      float w = device_attn[num_elems * head_idx + k];
      device_output_feats[global_idx] +=
          w * device_msgs[neighbor_idx * (num_heads * out_dim) + head_idx * out_dim + local_feat_idx];
    }
  }
}

void gatForwardCUDA(float *W, float *A, float *input_feats, sparse_matrix *adj, int in_dim,
                    int out_dim, int num_heads, int num_nodes, float *output_feats, float min_f) {
  float *device_W;
  float *device_input_feats;
  float *device_msgs;
  double startTime = CycleTimer::currentSeconds();

  hipMalloc((void **) &device_W, in_dim * num_heads * out_dim * sizeof(float));
  hipMalloc((void **) &device_input_feats, num_nodes * in_dim * sizeof(float));
  hipMalloc((void **) &device_msgs, num_nodes * num_heads * out_dim * sizeof(float));


  hipMemcpy(device_W, W, in_dim * num_heads * out_dim * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_input_feats, input_feats, num_nodes * in_dim * sizeof(float), hipMemcpyHostToDevice);

  dim3 threadsPerBlock = dim3(TILE_WIDTH, TILE_WIDTH);
  dim3 blocks = dim3((num_nodes + TILE_WIDTH - 1) / TILE_WIDTH, out_dim * num_heads + TILE_WIDTH - 1 / TILE_WIDTH);
  mm_kernel<<<blocks, threadsPerBlock>>>(num_nodes, in_dim, num_heads * out_dim, device_input_feats, device_W,
                                         device_msgs);
  hipDeviceSynchronize();
//  double endTime = CycleTimer::currentSeconds();
//  double overallDuration = endTime - startTime;
////  printf("Kernel invocation: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));
//  printf("Overall: %.3f ms\n", 1000.f * overallDuration);
//  float *test_msgs;
//  test_msgs = new float[num_nodes * num_heads * out_dim];
//  hipMemcpy(test_msgs, device_msgs, num_nodes * num_heads * out_dim * sizeof(float), hipMemcpyDeviceToHost);
//  printf("msgs %.3f %.3f %.3f %.3f\n", test_msgs[0], test_msgs[1], test_msgs[2], test_msgs[3]);
//  delete(test_msgs);
  hipFree(device_W);
  hipFree(device_input_feats);

  float *device_A;
  float *device_heats;
  hipMalloc((void **) &device_A, num_heads * 2 * out_dim * sizeof(float));
  hipMalloc((void **) &device_heats, num_heads * 2 * num_nodes * sizeof(float));

  hipMemcpy(device_A, A, num_heads * 2 * out_dim * sizeof(float), hipMemcpyHostToDevice);

  threadsPerBlock = dim3(2, TILE_WIDTH);
  blocks = dim3(num_heads, (num_nodes + TILE_WIDTH - 1) / TILE_WIDTH);

  heat_kernel<<<blocks, threadsPerBlock, 2 * out_dim * sizeof(float)>>>(num_nodes, num_heads, out_dim, device_msgs,
                                                                        device_A, device_heats);
  hipDeviceSynchronize();

//  float *test_heats;
//  test_heats = new float[num_heads * 2 * num_nodes];
//  hipMemcpy(test_heats, device_heats, num_heads * 2 * num_nodes * sizeof(float), hipMemcpyDeviceToHost);
//  printf("heats %.3f %.3f %.3f %.3f\n", test_heats[0], test_heats[1], test_heats[2], test_heats[3]);
//  delete(test_heats);

  hipFree(device_A);

  float *device_attn;
  int *device_col_idx;
  int *device_delim;
  hipMalloc((void **) &device_attn, num_heads * adj->num_elements * sizeof(float));
  hipMalloc((void **) &device_col_idx, adj->num_elements * sizeof(int));
  hipMalloc((void **) &device_delim, (adj->num_rows + 1) * sizeof(int));

  hipMemcpy(device_col_idx, adj->col_idx, adj->num_elements * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_delim, adj->delim, (adj->num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);


  threadsPerBlock = dim3(TILE_SIZE);
  blocks = dim3(num_heads, (num_nodes + TILE_SIZE - 1) / TILE_SIZE);

  attn_kernel<<<blocks, threadsPerBlock>>>(num_nodes, adj->num_elements,
                                           device_col_idx, device_delim, device_heats, device_attn, min_f);
  hipDeviceSynchronize();


//  float *test_attn;
//  test_attn = new float[num_heads * adj->num_elements];
//  hipMemcpy(test_attn, device_attn, num_heads * adj->num_elements * sizeof(float), hipMemcpyDeviceToHost);
//  printf("attn %.3f %.3f %.3f %.3f\n", test_attn[0], test_attn[1], test_attn[2], test_attn[3]);
//  delete(test_attn);
  hipFree(device_heats);

  float *device_output_feats;
  hipMalloc((void **) &device_output_feats, num_nodes * num_heads * out_dim * sizeof(float));

  threadsPerBlock = dim3(TILE_SIZE);
  blocks = dim3(num_heads, (num_nodes * out_dim + TILE_SIZE - 1) / TILE_SIZE);
  aggregate_kernel<<<blocks, threadsPerBlock>>>(num_nodes, num_heads, out_dim, adj->num_elements,
                                                device_msgs, device_attn, device_col_idx, device_delim,
                                                device_output_feats);
  hipDeviceSynchronize();

//  float *test_output_feats;
//  test_output_feats = new float[num_nodes * num_heads * out_dim];
//  hipMemcpy(test_output_feats, device_output_feats, num_nodes * num_heads * out_dim * sizeof(float), hipMemcpyDeviceToHost);
//  printf("output_feats %.3f %.3f %.3f %.3f\n", test_output_feats[0], test_output_feats[1], test_output_feats[2], test_output_feats[3]);
//  delete(test_output_feats);

  hipFree(device_msgs);
  hipFree(device_col_idx);
  hipFree(device_delim);

  hipMemcpy(output_feats, device_output_feats, num_heads * out_dim * num_nodes * sizeof(float),
             hipMemcpyDeviceToHost);
  hipFree(device_output_feats);

  double endTime = CycleTimer::currentSeconds();

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
  }

  double overallDuration = endTime - startTime;
//  printf("Kernel invocation: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));
  printf("Overall: %.3f ms\n", 1000.f * overallDuration);
}


void printCudaInfo() {
  // For fun, just print out some stats on the machine

  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}
